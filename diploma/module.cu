#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/device_functions.h>
#include <time.h>
#include <stdio.h>
#include <float.h>

extern "C"
{	
	struct Point {
		float x, y;
		int id;

		__device__ bool equal(Point &p) {
			return p.x==x && p.y==y;
		}
	};
	struct Line {
		Point p1;
		Point p2;
	};
	struct Polygon {
		Line *lines;
		int lines_count;
	};

enum {LEFT,  RIGHT,  BEYOND,  BEHIND, BETWEEN, ORIGIN, DESTINATION};
enum { INSIDE, OUTSIDE, BOUNDARY };   
enum { TOUCHING, CROSSING, INESSENTIAL }; 

	__device__  float gr_viz[100000000];//матрица смежности графа видимости размером point_len x point_len
	__device__  float gr_viz1[100000000];
	__device__  float distance[10000];
	__device__  bool visited[10000];

	__device__  bool mVisited[10000];
	__device__  float uDistance[10000];
	__device__  float cDistance[10000];

	__device__ Point *points;
	__device__ Polygon *polygons;
	__device__ int points_count;
	__device__ int polygons_count;
	__device__ int dimension;
	__device__ int k;

	__device__ __constant__ float MAX_VALUE = 1000000;
	
	__device__ __inline__ int pointInPolygon(Point &a);
	__device__ __inline__ bool isInsidePolygons2(float x1, float y1);
	__device__ __inline__ bool isInsidePolygons(Point &p);
	__device__ __inline__ bool isVisibleLine3(Point &p1, Point &p2);
	__device__ __inline__ bool isVisibleLine2(float &x1, float &y1, float &x2, float &y2);
	__device__ __inline__ bool isVisibleLine(Point &p1, Point &p2);
	__device__ __inline__ float evklid2(float x1, float y1, float x2, float y2);
	__device__ __inline__ float evklid(Point &p1, Point &p2);
	__device__ __inline__ bool intersect2(float &x1, float &y1, float &x2, float &y2,
										float &x3, float &y3, float &x4, float &y4);
	__device__ bool intersect(Point &a, Point &b, Point &c, Point &d);

	__global__ void deinit() {
		delete []points;
		delete []polygons[0].lines;
		delete []polygons;
	}

	__global__ void init(float *new_x, float *new_y, int *count, 
							int *p, int *p_len, int *d) {
		k = 0;
		dimension = d[0];
		points_count = count[0];
		points = new Point[points_count];
		polygons_count = p_len[0];
		polygons = new Polygon[polygons_count];
		

		for (int i = 0; i < points_count; i++) {
			points[i].x = new_x[i];
			points[i].y = new_y[i];
			points[i].id = i;

			distance[i] = MAX_VALUE;
			uDistance[i] = MAX_VALUE;
			cDistance[i] = MAX_VALUE;
			visited[i] = false;
			mVisited[i] = false;
		}

		uDistance[points_count-1] = 0;
		cDistance[points_count-1] = 0;
		mVisited[points_count-1] = true;

		int prev = 0;
		for (int i = 0; i < p_len[0]; i++) {
			int cur = p[i];
			int size = cur - prev;
			Line *lines = new Line[size];
			for (int j = 0; j < size; j++) {
				lines[j].p1 = points[prev + j];
				lines[j].p2 = points[prev + (j + 1) % size];
				//printf("	line %d, %d\n", prev + j, prev + (j + 1) % size);
			}

			polygons[i].lines = lines;
			polygons[i].lines_count = size;
			prev = p[i];
		}
	}

	__global__ void get_graph_viz(float *graph)
	{
		for (int i = 0; i < points_count; i++)
			for (int j=0; j < points_count; j++)
				graph[i*points_count+j] = gr_viz[i*points_count+j];
		//memcpy(graph, gr_viz, sizeof(float)*points_count*points_count);
	}

	__global__ void graph_viz()
	{
		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		int ind = i*points_count + j;
		int ind1 = j*points_count + i;

		if (i<j||ind >= points_count*points_count) {
			//printf("return %d, %d\n", i, j);
			return;
		}

		Point p1 = points[i];
		Point p2 = points[j];
		Point p;// = new Point();
		p.x = (p2.x + p1.x) / 2.0;
		p.y = (p2.y + p1.y) / 2.0;

		if (abs(p1.id - p2.id) <= 1) {
			gr_viz[ind1] = gr_viz1[ind1] = 
			gr_viz[ind] = gr_viz1[ind] = evklid(p1, p2);
		} else if (isVisibleLine3(p1, p2)) {
			gr_viz[ind1] = gr_viz1[ind1] = 
			gr_viz[ind] = gr_viz1[ind] = evklid(p1, p2);
		} else {
			gr_viz[ind1] = gr_viz1[ind1] = 
			gr_viz[ind] = gr_viz1[ind] = -1.0;
		}
//&& isInsidePolygons2((p2.x + p1.x) / 2.0, (p2.y + p1.y) / 2.0) && isInsidePolygons2((p2.x + 0.2*p1.x) / (1.0+0.2), (p2.y + 0.2*p1.y) / (1+0.2))
		//delete p;
	}

	__global__ void graph_viz1()
	{
		for (int i = 0; i < points_count; i++)
			for (int j = 0; j < points_count; j++) {
				int ind = i*points_count + j;
				int ind1 = j*points_count + i;

				if (i<j||ind >= points_count*points_count) {
					//printf("return %d, %d\n", i, j);
					continue;
				}

				Point p1 = points[i];
				Point p2 = points[j];
				Point p;// = new Point();
				p.x = (p2.x + p1.x) / 2.0;
				p.y = (p2.y + p1.y) / 2.0;

				if (abs(p1.id - p2.id) <= 1) {
					gr_viz[ind1] = gr_viz1[ind1] = 
					gr_viz[ind] = gr_viz1[ind] = evklid(p1, p2);
				} else if (isVisibleLine3(p1, p2)) {
					gr_viz[ind1] = gr_viz1[ind1] = 
					gr_viz[ind] = gr_viz1[ind] = evklid(p1, p2);
				} else {
					gr_viz[ind1] = gr_viz1[ind1] = 
					gr_viz[ind] = gr_viz1[ind] = MAX_VALUE;
				}
			}
//&& isInsidePolygons2((p2.x + p1.x) / 2.0, (p2.y + p1.y) / 2.0) && isInsidePolygons2((p2.x + 0.2*p1.x) / (1.0+0.2), (p2.y + 0.2*p1.y) / (1+0.2))
		//delete p;
	}

	__global__ void dijkstra1() {
		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= points_count)
			return;
		if (mVisited[i]) {
			mVisited[i] = false;

			for (int j = 0; j < points_count; j++) {
				int index = i*points_count+j;
				if (j >= points_count)
					continue;
				if (gr_viz[index] > 0.0) {
					if (uDistance[i] > cDistance[j] + gr_viz[index])
						uDistance[i] = cDistance[j] + gr_viz[index];
				}
			}
		}
	}


	__global__ void dijkstra2() {
		unsigned int ind = blockIdx.x * blockDim.x + threadIdx.x;

		if (cDistance[ind] >= uDistance[ind]) {
			cDistance[ind] = uDistance[ind];
			mVisited[ind] = true;
		} 

		uDistance[ind] = cDistance[ind];
	}

	__global__ void isEmpty(float *check) {
		check[0] = 0;
		for (int i = 0; i < points_count; i++)
			if (uDistance[i] > distance[i]) {
				check[0] = 1;
				return;
			}
	}

	__global__ void dPrint() {
		printf("\nСтоимость пути из начальной вершины до остальных: \n");
		for (int i=0; i<points_count; i++) 
			printf("%d > %d = %f\n", points_count-1, i, uDistance[i]-distance[i]);
	}

	__global__ void dijkstra(int *index_from, float *distances) {
		int st = points_count - 1, index, u;
		float min;
		index_from[st] = st;
		distances[st] = distance[st] = 0;

		for (int i = 0; i < points_count - 1; i++) {
			min = MAX_VALUE;
			for (int j = 0; j < points_count; j++) 
				if (!visited[j] && distance[j] <= min) {
					min = distance[j];
					index = j;
				}
			u = index;
			visited[u] = true;
			for (int j = 0; j < points_count; j++) 
				if (!visited[j] && gr_viz[u*points_count+j]>0.1 && distance[u]<MAX_VALUE && (distance[u]+gr_viz[u*points_count+j] < distance[j])) {
					index_from[j] = u;
					distances[j] = distance[j] = distance[u] + gr_viz[u*points_count+j];
				}
		}

		printf("\nСтоимость пути из начальной вершины до остальных: \n");
		for (int i=0; i<points_count; i++) 
			if (distance[i] >= 0 || distance[i] < MAX_VALUE)
				printf("%d > %d = %f\n", st, i, distance[i]);
			else 
				printf("%d > %d = маршрут не доступен\n", st, i);

	}

	__global__ void spm1(float *indexes) {

		for (int i = 0; i < dimension; i++) {
			for (int j = 0; j < dimension; j++) {
				unsigned int index = i*dimension+j;
				float _x = (float)i;
				float _y = (float)j;
				float min_dis = MAX_VALUE;
				float value_dis;

				if(!isInsidePolygons2(_x, _y)) {
					indexes[index] = 0;
					continue;
				}

				for (int k = 0; k < points_count; k++) {					
					value_dis = distance[k] + evklid2(points[k].x, points[k].y, _x, _y);
					if (value_dis < min_dis && isVisibleLine2(_x, _y, points[k].x, points[k].y)) {
						min_dis = value_dis;
						indexes[index] = (float)points[k].id;
					}
				}
			}
		}
	}

	__global__ void spm(float *indexes) {
		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		unsigned int index = i*dimension+j;
		if (index >= dimension*dimension) {
			//printf("return %d, %d\n", i, j);
			return;
		}
		float _x = (float)i;
		float _y = (float)j;
		if(!isInsidePolygons2(_x, _y)) {
			indexes[i*dimension+j] = 0;
			return;
		}

		float min_dis = MAX_VALUE;
		float value_dis;
		for (int k = 0; k < points_count; k++) {					
			value_dis = distance[k] + evklid2(points[k].x, points[k].y, _x, _y);
			if (value_dis < min_dis && isVisibleLine2(_x, _y, points[k].x, points[k].y)) {
				min_dis = value_dis;
				indexes[index] = (float)points[k].id;
			}
		}
	}

	__global__ void floyd2() {

		unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;

		if (i*points_count+j >= points_count*points_count)
			return;;
		float ij=gr_viz1[i*points_count+j], 
			ik=gr_viz1[i*points_count+k], 
			kj=gr_viz1[k*points_count+j];
		float result = 0;
		if (ik < 0 || kj < 0) 
			return;

		if (ij < 0)
			result = ik+kj;
		else
			result = min( ij, ik+kj );

		gr_viz1[i*points_count+j] = result;
		return;
		if (i == 0 && j == 0) {
			k++;
		}
	}

	__global__ void get_floyd2_result(float *matrix) {
		//unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		//unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		//memcpy(matrix, gr_viz1, sizeof(float)*points_count*points_count);
		for (int i = 0; i < points_count; i++)
			for (int j = 0; j < points_count; j++)
				matrix[i*points_count+j] = gr_viz1[i*points_count+j];
	}

	__global__ void floyd() {
		int index = (int)(((float)points_count / (float)min(32,points_count)) + 1);
		unsigned int index_x = index * threadIdx.x;
		unsigned int index_y = index * threadIdx.y;

		for (int k = 0; k < points_count; k++) {
					for (int i = index_x; i < index_x + index && i < points_count; i++) {
						for (int j = index_y; j < index_y + index && j < points_count; j++) {
							if (i*points_count+j >= points_count*points_count)
								continue;
							float ij=gr_viz[i*points_count+j], 
								ik=gr_viz[i*points_count+k], 
								kj=gr_viz[k*points_count+j];
							//matrix[i*points_count+j]=ij;
							float result = 0;
							if (ik < 0 || kj < 0) 
								continue;

							if (ij < 0)
								result = ik+kj;
							else
								result = min( ij, ik+kj );
							//matrix[i*points_count+j]=
							gr_viz[i*points_count+j] = result;
						}
					}
					__syncthreads();
		}
	}

	__global__ void get_floyd_result(float *matrix) {
		//unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		//unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		//memcpy(matrix, gr_viz, sizeof(float)*points_count*points_count);
		for (int i = 0; i < points_count; i++)
			for (int j = 0; j < points_count; j++)
				matrix[i*points_count+j] = gr_viz[i*points_count+j];
	}

	__global__ void floyd1(float *matrix) {
		//unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
		//unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
		//unsigned int index = i*points_count+j;
		
		for (int k = 0; k < points_count; k++)
			for (int i = 0; i < points_count; i++)
				for (int j = 0; j < points_count; j++) {
					float result = 0;
					if (gr_viz1[i*points_count+k] < 0 || gr_viz1[k*points_count+j] < 0) 
						continue;

					if (gr_viz1[i*points_count+j] < 0)
						result = gr_viz[i*points_count+k]+gr_viz1[k*points_count+j];
					else
						result = min( gr_viz1[i*points_count+j], gr_viz1[i*points_count+k]+gr_viz1[k*points_count+j]);
					gr_viz1[i*points_count+j] = matrix[i*points_count+j] = result;
				}
	}

	__device__ __inline__ bool pointInSegment(Point &t, Point &p1, Point &p2) {
		float k1 = (p1.x + p2.x) / p1.x;
		float k2 = (p1.x + p2.x) / p2.x;
		if (t.x < min(k1, k2) || t.x > max(k1, k2))
			return false;
		k1 = (p1.y + p2.y) / p1.y;
		k2 = (p1.y + p2.y) / p2.y;
		if (t.y < min(k1, k2) || t.y > max(k1, k2))
			return false;
		return true;
	}

	__device__ __inline__ bool pointInSegment2(float x3, float y3, float x1, float y1, float x2, float y2) {
		float k1 = (x1 + x2) / x1;
		float k2 = (x1 + x2) / x2;
		if (x3 < min(k1, k2) || x3 > max(k1, k2))
			return false;
		k1 = (y1 + y2) / y1;
		k2 = (y1 + y2) / y2;
		if (y3 < min(k1, k2) || y3 > max(k1, k2))
			return false;
		return true;
	}

	__device__ __inline__ bool isInsidePolygons2(float x1, float y1) {
		int inter_count = 0;
		float x2 = 0;
		float y2 = y1;

		for (int k = 0; k < polygons_count; k++) {
			Polygon polygon = polygons[k];
			inter_count = 0;
			for(int l = 0; l < polygon.lines_count; l++) {
				Line line = polygon.lines[l];					
				if (intersect2(x2, y2, x1, y1, line.p1.x, line.p1.y, line.p2.x, line.p2.y) )
					inter_count = 1 - inter_count;
				else if (pointInSegment2(x1, y1, line.p1.x, line.p1.y, line.p2.x, line.p2.y))
					return true;
			}
			if (inter_count == 1) {
				return true;
			}
		}
		return inter_count == 1;
	}

	__device__ __inline__ bool isInsidePolygons(Point &p1) {
		int inter_count = 0;
		Point *p2 = new Point();
		p2->x = 0;
		p2->y = p1.y;

		for (int k = 0; k < polygons_count; k++) {
			Polygon polygon = polygons[k];
			inter_count = 0;
			for(int l = 0; l < polygon.lines_count; l++) {
				Line line = polygon.lines[l];
				if (intersect(*p2, p1, line.p1, line.p2))
					inter_count = 1 - inter_count;
				else if (pointInSegment(p1, line.p1, line.p2))
					return true;
			}
			if (inter_count == 1) {
				delete p2;
				return true;
			}
		}
		delete p2;
		return inter_count == 1;
	}


	__device__ __inline__ bool isVisibleLine(Point &p1, Point &p2) {
		int diff_id = abs(p1.id - p2.id);
		for (int k = 0; k < polygons_count; k++) {
			Polygon polygon = polygons[k];
			if (diff_id > 1 && diff_id != polygon.lines_count - 1) {
				int id1 = p1.id - polygon.lines[0].p1.id;
				int id2 = p2.id - polygon.lines[0].p1.id;
				if ( id1 >= 0 && id2 >= 0 && id1 < polygon.lines_count && id2 < polygon.lines_count) {
					return false;
				}
			}

			for(int l = 0; l < polygon.lines_count; l++) {
				Line line = polygon.lines[l];
				if (p1.id!=line.p1.id && p2.id!=line.p1.id && p1.id!=line.p2.id && p2.id!=line.p2.id && 
					intersect(p1, p2, line.p1, line.p2)) {
						return false;
				}
			}
		}	
		return true;
	}

	__device__ __inline__ bool isVisibleLine2(float &x1, float &y1, float &x2, float &y2) {
		for (int k = 0; k < polygons_count; k++) {
			Polygon polygon = polygons[k];
			
			for(int l = 0; l < polygon.lines_count; l++) {
				Line line = polygon.lines[l];
				Point p3 = line.p1;
				Point p4 = line.p2;
				if (intersect2(x1, y1, x2, y2, p3.x, p3.y, p4.x, p4.y) && 
					!((x2 == p3.x && y2 == p3.y) || (x2 == p4.x && y2 == p4.y))) {
						return false;
				}
			}
		}	
		return true;
	}


	__device__ __inline__ bool leftSide(Point &p2, Point &p0, Point &p1)
	{
  		float sa = (p1.x - p0.x)*(p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y) ;
    	return sa > 0.0 ;
	}

	__device__ __inline__ int rightSide(Point &p2, Point &p0, Point &p1)
	{
  		float sa = (p1.x - p0.x)*(p2.y-p0.y)-(p1.y-p0.y)*(p2.x-p0.x); //(p1.x - p0.x)*(p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y) ;
    	return sa <= 0.0 ;
	}

	__device__ __inline__ bool isVisibleLine3( Point &p1, Point &p2 ) {

		int firstPoint = 0;
		int secondPoint = 0;
		Line l1, l2, l3, l4;
		for (int k = 0; k < polygons_count; k++) {
			Polygon polygon = polygons[k];
			
			for(int l = 0; l < polygon.lines_count; l++) {
				Line line = polygon.lines[l];
				Point p3 = line.p1;
				Point p4 = line.p2;
				if (intersect2(p1.x, p1.y, p2.x, p2.y, p3.x, p3.y, p4.x, p4.y) && 
					!((p2.id == p3.id) || (p2.id == p4.id) || (p1.id == p3.id) || (p1.id == p4.id))) {
						return false;
				}
				if ((p1.id == p3.id || p1.id == p4.id) && leftSide(p2, p3, p4)) {
					firstPoint++;
				}
				if ((p2.id == p3.id || p2.id == p4.id) && leftSide(p1, p3, p4)) {
					secondPoint++;
				}
						
			}
		}	
		if (firstPoint + secondPoint < 2)
			return true;
		float ratio = 11.0;
		float ratio1 = 0.1;
		return isInsidePolygons2((p2.x + ratio*p1.x) / (1.0+ratio), (p2.y + ratio*p1.y) / (1.0+ratio))
		&& isInsidePolygons2((p2.x + ratio1*p1.x) / (1.0+ratio1), (p2.y + ratio1*p1.y) / (1.0+ratio1));
	}

	__device__ __inline__ float evklid2(float x1, float y1, float x2, float y2) {
		float m1 = x1 - x2;
		float m2 = y1 - y2;
		return sqrt(m1*m1 + m2*m2);
	}

	__device__ __inline__ float evklid(Point &p1, Point &p2) {
		return sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2));
	}

	__device__ __inline__ void swap(float &a, float &b) {
		float c = a;
		a = b;
		b = c;
	}

	__inline__ __device__ float area2 (float &x1, float &y1, float &x2, float &y2, float &x3, float &y3) {
		return (x2 - x1) * (y3 - y1) - (y2 - y1) * (x3 - x1);
	}
	__inline__ __device__ float area (Point &a, Point &b, Point &c) {
		return (b.x - a.x) * (c.y - a.y) - (b.y - a.y) * (c.x - a.x);
	}
	 
	__inline__ __device__ bool intersect_1 (float a, float b, float c, float d) {
		if (a > b)  swap (a, b);
		if (c > d)  swap (c, d);
		return max(a,c) <= min(b,d);
	}

	__device__ __inline__ bool intersect2(float &x1, float &y1, float &x2, float &y2,
										float &x3, float &y3, float &x4, float &y4) {
		return 
			area2(x1, y1, x2, y2, x3, y3) * area2(x1, y1, x2, y2, x4, y4) <= 0
			&& area2(x3, y3, x4, y4, x1, y1) * area2(x3, y3, x4, y4, x2, y2) <= 0
			&& intersect_1 (x1, x2, x3, x4)
			&& intersect_1 (y1, y2, y3, y4);
	}
	 
	__device__ __inline__ bool intersect (Point &a, Point &b, Point &c, Point &d) {
		return intersect_1 (a.x, b.x, c.x, d.x)
			&& intersect_1 (a.y, b.y, c.y, d.y)
			&& area(a,b,c) * area(a,b,d) <= 0
			&& area(c,d,a) * area(c,d,b) <= 0;
	}



__device__ __inline__ int classify(Point &p2, Point &p0, Point &p1)
{
  float sa = (p1.x - p0.x)*(p2.y - p0.y) - (p2.x - p0.x) * (p1.y - p0.y);
  if (sa > 0.0)
    return LEFT;
  if (sa < 0.0)
    return RIGHT;
  if (((p1.x - p0.x) * (p2.x - p0.x)< 0.0) || ((p1.y - p0.y) * (p2.y - p0.y) < 0.0))
    return BEHIND;
  if (evklid2(p1.x - p0.x, p1.y - p0.y, p1.x - p0.x, p1.y - p0.y) < evklid2(p2.x - p0.x, p2.y - p0.y, p2.x - p0.x, p2.y - p0.y))
    return BEYOND;
  if (fabs(p0.x - p2.x) < 0.01 && fabs(p0.y - p2.y) < 0.01)
    return ORIGIN;
  if (fabs(p1.x - p2.x) < 0.01 && fabs(p1.y - p2.y) < 0.01)
    return DESTINATION;
  return BETWEEN;
}

__device__ __inline__ int edgeType(Point &a, Line &e)
{
  Point v = e.p1;
  Point w = e.p2;
  switch (classify(a, e.p1, e.p2)) {
    case LEFT:
      return ((v.y<a.y)&&(a.y<=w.y)) ? CROSSING : INESSENTIAL; 
    case RIGHT:
      return ((w.y<a.y)&&(a.y<=v.y)) ? CROSSING : INESSENTIAL; 
    case BETWEEN:
    case ORIGIN:
    case DESTINATION:
      return TOUCHING;
    default:
      return INESSENTIAL;
  }
}


__device__ __inline__ int pointInPolygon(Point &a)
{
	int parity = 0;
	Polygon polygon = polygons[0];
	for(int l = 0; l < polygon.lines_count; l++) {
		Line line = polygon.lines[l];
		switch (edgeType(a, line)) {
    		case TOUCHING:
    			return BOUNDARY;
    		case CROSSING:
    			parity = 1 - parity;
		}
	}

	return (parity ? INSIDE : OUTSIDE);
}
}
